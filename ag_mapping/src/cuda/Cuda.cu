#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace AG_MAPPING { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
